#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * This sample implements a conjugate gradient solver on GPU
 * using CUBLAS and CUSPARSE
 *
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <windows.h>

/* Using updated (v2) interfaces to cublas */
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>

// Utilities and system includes
#include <helper_functions.h>  // helper for shared functions common to CUDA Samples
#include <hip/hip_runtime_api.h>       // helper function CUDA error checking and initialization

#include <string>

const char *sSDKname     = "conjugateGradient";
extern "C"
float* gc( float*,int,int,int);
class Clock {
private:
	static const int N = 50;
	int index;
	LARGE_INTEGER startCounter[N], frequency, stopCounter[N];
	std::string text;
public:
	Clock() {
		QueryPerformanceFrequency(&frequency);
		text = "";
		index = 0;
	}
	Clock(std::string text) :text(text) {
		QueryPerformanceFrequency(&frequency);
		index = 0;
	}
	void start() {
		QueryPerformanceCounter(&startCounter[++index]);
	}
	void end(std::string text) {
		QueryPerformanceCounter(&stopCounter[index]);
		printf("%s %f\n", text.c_str(), (double)(stopCounter[index].QuadPart - startCounter[index].QuadPart) / frequency.QuadPart);
		index--;
	}
	void end() {
		QueryPerformanceCounter(&stopCounter[index]);
		printf("%s %f\n", text.c_str(), (double)(stopCounter[index].QuadPart - startCounter[index].QuadPart) / frequency.QuadPart);
		index--;
	}
};
// I[i] ��I�е�һ����0����val�е�index
/* genTridiag: generate a random tridiagonal symmetric matrix */
void gen(int *I, int *J, float *val, int N, int nz,int X,int Y,int Z)
{
	Clock clock;
	clock.start();
    I[0] = 0, J[0] = 0, J[1] = 1, J[2] = Y, J[3] = Y*Z;
	val[0] = 6;
	val[1] = -1;
	val[2] = -1;
	val[3] = -1;
    int start;
	int k = 4;
	int count = X*Y*Z;
    for (int i = 1; i < N; i++)
    {
		I[i] = k;
		if (i - Y*Z + 1>0) {
			J[k] = i - Y*Z;
			val[k] = -1;
			k++;
		}
		if (i - Y + 1 > 0) {
			J[k] = i - Y;
			val[k] = -1;
			k++;
		}
		if (i > 0) {
			J[k] = i - 1;
			val[k] = -1;
			k++;
		}
		J[k] = i;
		val[k] = 6;
		k++;
		if (i + 1 < count) {
			J[k] = i + 1;
			val[k] = -1;
			k++;
		}
		if (i + Y < count) {
			J[k] = i + Y;
			val[k] = -1;
			k++;
		}
		if (i + Y*Z < count) {
			J[k] = i + Y*Z;
			val[k] = -1;
			k++;
		}
    }

    I[N] = k;
	clock.end("prepare CSR A");
}

extern "C"
float* gc(float* data,int X,int Y,int Z )
{
	Clock clock;
	static bool flag = true;
    static int M = 0, N = 0, nz = 0, *I = NULL, *J = NULL;
    float *val = NULL;
    const float tol = 1e-5f;
    const int max_iter = 1;
    float *x;
    float *rhs;
    float a, b, na, r0, r1;
    int *d_col, *d_row;
    float *d_val, *d_x, dot;
    float *d_r, *d_p, *d_Ax;
    int k;
    float alpha, beta, alpham1;
	int count = X*Y*Z;
    /* Generate a random tridiagonal symmetric matrix in CSR format */
    M = N = count;
	clock.start();
	clock.start();
	if (flag) {
		for (int i = 0; i < N; i++)
		{
			if (i - Y*Z + 1 > 0) {
				nz++;
			}
			if (i - Y + 1 > 0) {
				nz++;
			}
			if (i > 0) {
				nz++;
			}
			nz++;
			if (i + 1 < count) {
				nz++;
			}
			if (i + Y < count) {
				nz++;
			}
			if (i + Y*Z < count) {
				nz++;
			}
		}
		clock.end("count total");
		I = (int *)malloc(sizeof(int)*(N + 1));
		J = (int *)malloc(sizeof(int)*nz);
		val = (float *)malloc(sizeof(float)*nz);
		gen(I, J, val, N, nz, X, Y, Z);
	}

    x = (float *)malloc(sizeof(float)*N);
    rhs = (float *)malloc(sizeof(float)*N);
	clock.start();
    for (int i = 0; i < N; i++)
    {
        rhs[i] = data[i];
        x[i] = 0.0;
    }

    /* Get handle to the CUBLAS context */
    hipblasHandle_t cublasHandle = 0;
    hipblasStatus_t hipblasStatus_t;
    hipblasStatus_t = hipblasCreate(&cublasHandle);

    checkCudaErrors(hipblasStatus_t);

    /* Get handle to the CUSPARSE context */
    hipsparseHandle_t cusparseHandle = 0;
    hipsparseStatus_t cusparseStatus;
    cusparseStatus = hipsparseCreate(&cusparseHandle);

    checkCudaErrors(cusparseStatus);

    hipsparseMatDescr_t descr = 0;
    cusparseStatus = hipsparseCreateMatDescr(&descr);

    checkCudaErrors(cusparseStatus);

    hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);

    checkCudaErrors(hipMalloc((void **)&d_col, nz*sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&d_row, (N+1)*sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&d_val, nz*sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_x, N*sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_r, N*sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_p, N*sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_Ax, N*sizeof(float)));

    hipMemcpy(d_col, J, nz*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_row, I, (N+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_val, val, nz*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_r, rhs, N*sizeof(float), hipMemcpyHostToDevice);

    alpha = 1.0;
    alpham1 = -1.0;
    beta = 0.0;
    r0 = 0.;
    hipsparseScsrmv(cusparseHandle,HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nz, &alpha, descr, d_val, d_row, d_col, d_x, &beta, d_Ax);
    hipblasSaxpy(cublasHandle, N, &alpham1, d_Ax, 1, d_r, 1);
    hipblasStatus_t = hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);

    k = 1;

    while (r1 > tol*tol && k <= max_iter)
    {
        if (k > 1)
        {
            b = r1 / r0;
            hipblasStatus_t = hipblasSscal(cublasHandle, N, &b, d_p, 1);
            hipblasStatus_t = hipblasSaxpy(cublasHandle, N, &alpha, d_r, 1, d_p, 1);
        }
        else
        {
            hipblasStatus_t = hipblasScopy(cublasHandle, N, d_r, 1, d_p, 1);
        }

        hipsparseScsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nz, &alpha, descr, d_val, d_row, d_col, d_p, &beta, d_Ax);
        hipblasStatus_t = hipblasSdot(cublasHandle, N, d_p, 1, d_Ax, 1, &dot);
        a = r1 / dot;

        hipblasStatus_t = hipblasSaxpy(cublasHandle, N, &a, d_p, 1, d_x, 1);
        na = -a;
        hipblasStatus_t = hipblasSaxpy(cublasHandle, N, &na, d_Ax, 1, d_r, 1);

        r0 = r1;
        hipblasStatus_t = hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);
        hipDeviceSynchronize();
        printf("iteration = %3d, residual = %e\n", k, sqrt(r1));
        k++;
    }

    hipMemcpy(x, d_x, N*sizeof(float), hipMemcpyDeviceToHost);
	clock.end("cal:");
	//error estimate
    /*
	clock.start();
	float rsum, diff, err = 0.0;

    for (int i = 0; i < N; i++)
    {
        rsum = 0.0;

        for (int j = I[i]; j < I[i+1]; j++)
        {
            rsum += val[j]*x[J[j]];
        }

        diff = fabs(rsum - rhs[i]);

        if (diff > err)
        {
            err = diff;
        }
    }
	clock.end("error");
	*/

    hipsparseDestroy(cusparseHandle);
    hipblasDestroy(cublasHandle);

    //free(I);
    //free(J);
    //free(val);
    free(rhs);
    hipFree(d_col);
    hipFree(d_row);
    hipFree(d_val);
    hipFree(d_x);
    hipFree(d_r);
    hipFree(d_p);
    hipFree(d_Ax);
	clock.end("total time:");
	// printf("Test Summary:  Error amount = %f\n", err);
    //exit((k <= max_iter) ? 0 : 1);
	return x;
}
