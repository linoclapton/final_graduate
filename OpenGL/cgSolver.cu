#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * This sample implements a conjugate gradient solver on GPU
 * using CUBLAS and CUSPARSE
 *
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <windows.h>

/* Using updated (v2) interfaces to cublas */
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>

// Utilities and system includes
#include <helper_functions.h>  // helper for shared functions common to CUDA Samples
#include <hip/hip_runtime_api.h>       // helper function CUDA error checking and initialization

#include <string>

const char *sSDKname     = "conjugateGradient";
extern "C"
float* gc( int X, int Y, int Z, float* i_udata, int *dims, int dim, float* i_opacity, float *i_min, float *i_max);


class DataBlock{
public:
	int *dimension;
	int *idx;
	int *DIM;
	float* opacity;
	float* udata;
	float *min;
	float *max;
	int *Nd;
	float *ux;
};


class Clock {
private:
	static const int N = 50;
	int index;
	LARGE_INTEGER startCounter[N], frequency, stopCounter[N];
	std::string text;
public:
	Clock() {
		QueryPerformanceFrequency(&frequency);
		text = "";
		index = 0;
	}
	Clock(std::string text) :text(text) {
		QueryPerformanceFrequency(&frequency);
		index = 0;
	}
	void start() {
		QueryPerformanceCounter(&startCounter[++index]);
	}
	void end(std::string text) {
		QueryPerformanceCounter(&stopCounter[index]);
		printf("%s %f\n", text.c_str(), (double)(stopCounter[index].QuadPart - startCounter[index].QuadPart) / frequency.QuadPart);
		index--;
	}
	void end() {
		QueryPerformanceCounter(&stopCounter[index]);
		printf("%s %f\n", text.c_str(), (double)(stopCounter[index].QuadPart - startCounter[index].QuadPart) / frequency.QuadPart);
		index--;
	}
};
// I[i] ��I�е�һ����0����val�е�index
/* genTridiag: generate a random tridiagonal symmetric matrix */
void gen(int *I, int *J, float *val, int N, int nz,int X,int Y,int Z)
{
	Clock clock;
	clock.start();
    I[0] = 0, J[0] = 0, J[1] = 1, J[2] = Y, J[3] = Y*Z;
	val[0] = 6;
	val[1] = -1;
	val[2] = -1;
	val[3] = -1;
    int start;
	int k = 4;
	int count = X*Y*Z;
    for (int i = 1; i < N; i++)
    {
		I[i] = k;
		if (i - Y*Z + 1>0) {
			J[k] = i - Y*Z;
			val[k] = -1;
			k++;
		}
		if (i - Y + 1 > 0) {
			J[k] = i - Y;
			val[k] = -1;
			k++;
		}
		if (i > 0) {
			J[k] = i - 1;
			val[k] = -1;
			k++;
		}
		J[k] = i;
		val[k] = 6;
		k++;
		if (i + 1 < count) {
			J[k] = i + 1;
			val[k] = -1;
			k++;
		}
		if (i + Y < count) {
			J[k] = i + Y;
			val[k] = -1;
			k++;
		}
		if (i + Y*Z < count) {
			J[k] = i + Y*Z;
			val[k] = -1;
			k++;
		}
    }

    I[N] = k;
	clock.end("prepare CSR A");
}
// dim 1 dimension 3 windowMin&Max 2 opacity 1 udata 1 Nx,Ny,Nz 
__device__ float getOpacity(float* opacity,float g) {
	return  opacity[int(g * 255)];
}
__device__ float index(DataBlock block,int x, int y, int z) {
	float* udata = block.udata;
	int DIM = block.DIM[0];
	int* dimension = block.dimension;
	float mx = block.max[0];
	float mn = block.min[0];
	float d = udata[DIM*x*dimension[1] * dimension[0] + DIM*y*dimension[0] + DIM*z];
	if (d < mn ) return 0.0;
	if (d > mx) return 1.0;

	return (d - mn) / (mx - mn);
}

__global__ void convection(float *p,DataBlock block) {
	int Nx = block.Nd[0];
	int Ny = block.Nd[1];
	int Nz = block.Nd[2];
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int z = threadIdx.z + blockIdx.z * blockDim.z;
	int offset = z + y*Nz + x*Ny*Nz;
	if (offset >= Nx*Ny*Nz) return;
	//float err = 1.0;
	//while(err>0.1) {
		/*if (p[offset] < 0.0) {
			p[offset] = 1.0 - getOpacity(block.opacity, index(block, x, y, z));
			return;
		}*/
	float op = getOpacity(block.opacity, index(block, x, y, z));
	for (int i = 0; i < 25; i++) {
		int right = offset + 1;
		int left = offset - 1;
		int down = offset + Nz; 
		int up = offset - Nz;
		int top = offset + Ny*Nz;
		int bottom = offset - Ny*Nz;
		float ux[] = { 0.577,0.577,0.577 };
		if (x == 0) {
			bottom += Ny*Nz;
		}
		if (x == (Nx - 1)) {
			top -= Ny*Nz;
		}
		if (y == 0) { up += Nz; }
		if (y == (Ny - 1)) { down -= Nz; }
		if (z == 0) { left++; }
		if (z == (Nz - 1)) { right--; }

		float flux_right = 0.0;
		float flux_left = 0.0;
		if (ux[0] < 0.) { flux_right = (p[right] - p[offset]) * ux[0]; }
		else { flux_left = (p[offset] - p[left])* ux[0]; }

		float flux_down = 0.0;
		float flux_up = 0.0;
		if (ux[1] < 0.) { flux_down = (p[down] - p[offset]) * ux[1]; }
		else { flux_up = (p[offset] - p[up])* ux[1]; }

		float flux_top;
		float flux_bottom;
		if (ux[2] < 0.) { flux_top = (p[top] - p[offset]) * ux[2]; }
		else { flux_bottom = (p[offset] - p[bottom])* ux[1]; }
		p[offset] = (1 - op)*(p[offset] - 0.5*(
			flux_right + flux_left + flux_up + flux_down + flux_top + flux_bottom));
		if (p[offset] > 1.0) p[offset] = 1.0;
		if (p[offset] < 0.001) p[offset] = 0.0;
	}
	//	err = -c*(ux[0] * (flux_right + flux_left) + ux[1] * (flux_down + flux_up) + ux[2] * (flux_top + flux_bottom)) - (1 - op)*p[offset];
	//}
}

static bool flag = true;
extern "C"
float* gc(int X, int Y, int Z, float* i_udata, int *dims, int dim, float* i_opacity, float *i_min, float *i_max)
{
	Clock clock;
	static int M = 0, N = 0, nz = 0, *I = NULL, *J = NULL;
	static float *init_bound = NULL;
    float *val = NULL;
    const float tol = 1e-5f;
    const int max_iter = 1;
    float *x;
    float *rhs;
    float a, b, na, r0, r1;
    static int *d_col, *d_row;
	static float *d_val;
	float *d_x, dot;
	static float *d_r;
	float *d_p, *d_Ax;
    int k;
    float alpha, beta, alpham1;
	int Nd[] = { X,Y,Z };
	int count = X*Y*Z;
	static DataBlock block;
	static int *dimension;
	static float *opacity;
	static float *udata;
    /* Generate a random tridiagonal symmetric matrix in CSR format */
    M = N = count;
	clock.start();
	clock.start();
	if (flag) {
		checkCudaErrors(hipMalloc(&(block.udata), sizeof(float)*dims[0] * dims[1] * dims[2]));
		checkCudaErrors(hipMalloc(&d_r, sizeof(float)*count));
		checkCudaErrors(hipMalloc(&(block.dimension), sizeof(int)*3));
		checkCudaErrors(hipMalloc(&(block.Nd), sizeof(int)*3));
		checkCudaErrors(hipMalloc(&(block.max), sizeof(float)*4));
		checkCudaErrors(hipMalloc(&(block.min), sizeof(float)*4));
		checkCudaErrors(hipMalloc(&(block.DIM), sizeof(int)));
		checkCudaErrors(hipMalloc(&(block.opacity), sizeof(float)*256));
		checkCudaErrors(hipMemcpy(block.udata, i_udata, sizeof(float)*dims[0] * dims[1] * dims[2],hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(block.dimension,dims, sizeof(int)*3,hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(block.Nd,Nd, sizeof(int)*3,hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(block.DIM,&dim, sizeof(int),hipMemcpyHostToDevice));
		//checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(&DIM), &dim, sizeof(int)));
		for (int i = 0; i < N; i++)
		{
			if (i - Y*Z + 1 > 0) {
				nz++;
			}
			if (i - Y + 1 > 0) {
				nz++;
			}
			if (i > 0) {
				nz++;
			}
			nz++;
			if (i + 1 < count) {
				nz++;
			}
			if (i + Y < count) {
				nz++;
			}
			if (i + Y*Z < count) {
				nz++;
			}
		}
		init_bound = (float*)malloc(sizeof(float)*N);
		for (int i = 0; i < X; i++)
			for (int j = 0; j < Y; j++)
				for (int k = 0; k < Z; k++)
					if (i == 0 || j == 0 || k == 0)
						init_bound[i*Y*Z + j*Z + k] = 1.0;
					else if (i == X - 1 || j == Y - 1 || k == Z - 1)
						init_bound[i*Y*Z + j*Z + k] = 0.0;
					else
						init_bound[i*Y*Z + j*Z + k] = 0.0;// 333 * (init_bound[(i - 1)*Y*Z + j*Z + k] + init_bound[i*Y*Z + (j - 1)*Z + k] + init_bound[i*Y*Z + j*Z + k - 1]);
		clock.end("count total");
		I = (int *)malloc(sizeof(int)*(N + 1));
		J = (int *)malloc(sizeof(int)*nz);
		val = (float *)malloc(sizeof(float)*nz);
		gen(I, J, val, N, nz, X, Y, Z);
	}
	checkCudaErrors(hipMemcpy(block.opacity,i_opacity, sizeof(float)*256,hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_r, init_bound,sizeof(float)*count,hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(block.min, i_min, sizeof(int),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(block.max, i_max, sizeof(int),hipMemcpyHostToDevice));
	//checkCudaErrors(hipMemcpy(block.idx, &i, sizeof(int),hipMemcpyHostToDevice));
	int i;
	int tb[] = { 4,8,8 };
	dim3 grid((X+tb[0]-1)/tb[0],(Y+tb[1]-1)/tb[1],(Z+tb[2]-1)/tb[2]);
	dim3 thread(tb[0],tb[1],tb[2]);
	for (i = 0; i < 2 ; i++) {
		convection << <grid, thread>> > ( d_r, block );
		hipDeviceSynchronize();
	}
	//rhs = (float*)malloc(sizeof(float)*N);
	//hipMemcpy(rhs, d_r, sizeof(float)*count, hipMemcpyDeviceToHost);
    x = (float *)malloc(sizeof(float)*N);
	clock.start();
    for (int i = 0; i < N; i++)
    {
        //rhs[i] = data[i];
        x[i] = 0.0;
    }

    /* Get handle to the CUBLAS context */
    hipblasHandle_t cublasHandle = 0;
    hipblasStatus_t hipblasStatus_t;
    hipblasStatus_t = hipblasCreate(&cublasHandle);

    checkCudaErrors(hipblasStatus_t);

    /* Get handle to the CUSPARSE context */
    hipsparseHandle_t cusparseHandle = 0;
    hipsparseStatus_t cusparseStatus;
    cusparseStatus = hipsparseCreate(&cusparseHandle);

    checkCudaErrors(cusparseStatus);

    hipsparseMatDescr_t descr = 0;
    cusparseStatus = hipsparseCreateMatDescr(&descr);

    checkCudaErrors(cusparseStatus);

    hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);

	if (flag) {
		checkCudaErrors(hipMalloc((void **)&d_col, nz * sizeof(int)));
		checkCudaErrors(hipMalloc((void **)&d_row, (N + 1) * sizeof(int)));
		checkCudaErrors(hipMalloc((void **)&d_val, nz * sizeof(float)));
		//checkCudaErrors(hipMalloc((void **)&d_r, N * sizeof(float)));
	}
	checkCudaErrors(hipMalloc((void **)&d_x, N * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_p, N*sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_Ax, N*sizeof(float)));

    hipMemcpy(d_col, J, nz*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_row, I, (N+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_val, val, nz*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
    //hipMemcpy(d_r, rhs, N*sizeof(float), hipMemcpyHostToDevice);

    alpha = 1.0;
    alpham1 = -1.0;
    beta = 0.0;
    r0 = 0.;
    hipsparseScsrmv(cusparseHandle,HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nz, &alpha, descr, d_val, d_row, d_col, d_x, &beta, d_Ax);
    hipblasSaxpy(cublasHandle, N, &alpham1, d_Ax, 1, d_r, 1);
    hipblasStatus_t = hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);

    k = 1;

    while (r1 > tol*tol && k <= max_iter)
    {
        if (k > 1)
        {
            b = r1 / r0;
            hipblasStatus_t = hipblasSscal(cublasHandle, N, &b, d_p, 1);
            hipblasStatus_t = hipblasSaxpy(cublasHandle, N, &alpha, d_r, 1, d_p, 1);
        }
        else
        {
            hipblasStatus_t = hipblasScopy(cublasHandle, N, d_r, 1, d_p, 1);
        }

        hipsparseScsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nz, &alpha, descr, d_val, d_row, d_col, d_p, &beta, d_Ax);
        hipblasStatus_t = hipblasSdot(cublasHandle, N, d_p, 1, d_Ax, 1, &dot);
        a = r1 / dot;

        hipblasStatus_t = hipblasSaxpy(cublasHandle, N, &a, d_p, 1, d_x, 1);
        na = -a;
        hipblasStatus_t = hipblasSaxpy(cublasHandle, N, &na, d_Ax, 1, d_r, 1);

        r0 = r1;
        hipblasStatus_t = hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);
        hipDeviceSynchronize();
        //printf("iteration = %3d, residual = %e\n", k, sqrt(r1));
        k++;
    }

    hipMemcpy(x, d_x, N*sizeof(float), hipMemcpyDeviceToHost);
	clock.end("cal:");
	//error estimate
    /*
	clock.start();
	float rsum, diff, err = 0.0;

    for (int i = 0; i < N; i++)
    {
        rsum = 0.0;

        for (int j = I[i]; j < I[i+1]; j++)
        {
            rsum += val[j]*x[J[j]];
        }

        diff = fabs(rsum - rhs[i]);

        if (diff > err)
        {
            err = diff;
        }
    }
	clock.end("error");
	*/

    hipsparseDestroy(cusparseHandle);
    hipblasDestroy(cublasHandle);

    //free(I);
    //free(J);
    //free(val);
    //free(rhs);
    //hipFree(d_col);
    //hipFree(d_row);
    //hipFree(d_val);
    hipFree(d_x);
    //hipFree(d_r);
    hipFree(d_p);
    hipFree(d_Ax);
	clock.end("total time:");
	if (flag) flag = false;
	// printf("Test Summary:  Error amount = %f\n", err);
    //exit((k <= max_iter) ? 0 : 1);
	return x;
}

/*int main() {
	float *udata = (float*)malloc(512 * 512 * 100 * sizeof(float));
	int dim[3] = { 512,512,100 };
	float *o= (float*)malloc(256* sizeof(float));
	gc(256, 256, 50, udata, dim,2,o, 0.0f, 1.0f);
}*/
